#include "hip/hip_runtime.h"
#include <gputk.h>

#define NUM_STREAMS 16

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  unsigned int numStreams;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(GPU, "Allocating Pinned memory.");

  //@@ Allocate GPU memory here using pinned memory here
  hipHostRegister(hostInput1, inputLength * sizeof(float),
                   hipHostRegisterMapped);
  hipHostRegister(hostInput2, inputLength * sizeof(float),
                   hipHostRegisterMapped);
  hipHostRegister(hostOutput, inputLength * sizeof(float),
                   hipHostRegisterMapped);

  hipMalloc(&deviceInput1, inputLength * sizeof(float));
  hipMalloc(&deviceInput2, inputLength * sizeof(float));
  hipMalloc(&deviceOutput, inputLength * sizeof(float));

  //@@ Create and setup streams
  numStreams = NUM_STREAMS;

  hipStream_t *streams =
      (hipStream_t *)malloc(numStreams * sizeof(hipStream_t));

  for (unsigned int s = 0; s < numStreams; s++) {
    hipStreamCreate(&(streams[s]));
  }

  //@@ Calculate data segment size of input data processed by each stream
  int segmentLength = ceil((float)inputLength / numStreams);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Perform parallel vector addition with different streams.
  for (unsigned int s = 0; s < numStreams; s++) {
    //@@ Asynchronous copy data to the device memory in segments
    int streamSegmentLength = std::max(
        0, std::min(inputLength - (int)s * segmentLength, segmentLength));

    hipMemcpyAsync(deviceInput1 + s * segmentLength,
                    hostInput1 + s * segmentLength,
                    streamSegmentLength * sizeof(float), hipMemcpyHostToDevice,
                    streams[s]);

    hipMemcpyAsync(deviceInput2 + s * segmentLength,
                    hostInput2 + s * segmentLength,
                    streamSegmentLength * sizeof(float), hipMemcpyHostToDevice,
                    streams[s]);

    //@@ Calculate starting and ending indices for per-stream data

    //@@ Invoke CUDA Kernel
    //@@ Determine grid and thread block sizes (consider ococupancy)
    dim3 blockDim(256);
    dim3 gridDim(ceil((float)streamSegmentLength / blockDim.x));

    vecAdd<<<gridDim, blockDim, 0, streams[s]>>>(
        deviceInput1 + s * segmentLength, deviceInput2 + s * segmentLength,
        deviceOutput + s * segmentLength, segmentLength);

    //@@ Asynchronous copy data from the device memory in segments
    hipMemcpyAsync(hostOutput + s * segmentLength,
                    deviceOutput + s * segmentLength,
                    streamSegmentLength * sizeof(float), hipMemcpyDeviceToHost,
                    streams[s]);
  }

  //@@ Synchronize
  for (unsigned int s = 0; s < numStreams; s++) {
    hipStreamSynchronize(streams[s]);
  }
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(GPU, "Freeing Pinned Memory");
  //@@ Destory cudaStream
  for (unsigned int s = 0; s < numStreams; s++) {
    hipStreamDestroy(streams[s]);
  }

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  hipHostUnregister(hostInput1);
  hipHostUnregister(hostInput2);
  hipHostUnregister(hostOutput);

  gpuTKTime_stop(GPU, "Freeing Pinned Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
